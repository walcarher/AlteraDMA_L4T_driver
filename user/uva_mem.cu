#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void init_value_device(float *a, const int N){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i < N) a[i] = 0.5f;

}

int main(int argc, char **argv) {
	// Set device	
	int dev = 0;
	hipSetDevice(dev);
	
	// Memory size
	unsigned int isize = 1<<7;
	unsigned int nbytes = isize *sizeof(float);
	
	// Get Device infromation and check CPU memory mapping 
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	if (!deviceProp.canMapHostMemory){
		printf("Device %d does not support CPU mapping CPU host memory\n", dev);
		hipDeviceReset();
		exit(EXIT_SUCCESS);	
	}
	printf("Device %d: %s memory size %d nbyte %5.2fMB\n", dev, deviceProp.name, isize,nbytes/(1024.0f*1024.0f));
	
	// Allocate the host memory 
	float *h_a = (float *)malloc(nbytes);

	// Allocate the device memory
	float *d_a;
	hipMalloc((float **) &d_a, nbytes);

	// Initialize host memory variable a
	for (unsigned int i = 0; i < isize; i++) h_a[i] = 0.5f;

	// Transfer from host to device 
	hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice); 

	// Transfer from device to host
	hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost); 
	
	// Free memory on host and device
	free(h_a);
	hipFree(d_a);
	hipDeviceReset();

	///////// 2nd Part ZERO-COPY MEMORY for integrated heterogeneous architecture ////////////
	
	float* h_b = NULL;	

	// Allocate data at host side with zero-copy
	hipHostAlloc((void **)&h_b, isize, hipHostMallocMapped);	

	float* d_b;

	// Pass pointer to device 
	hipHostGetDevicePointer((void **)&d_b, (void *)h_b, 0);

	// Initialize variable b on device 
	int iLen = 512;
	dim3 block (iLen);
	dim3 grid ((isize+block.x-1)/block.x);
	init_value_device<<<grid,block>>>(d_b, nbytes);

	hipMemcpy(h_b, d_b, nbytes, hipMemcpyDeviceToHost);
	
	for (unsigned int i = 0; i < isize; i++) printf("%1.2f\n",h_b[i]);

	// Free memory on host and device
	hipHostFree(h_b);
	hipFree(d_b);

	// Reset device
	hipDeviceReset();	
	return EXIT_SUCCESS;

}
